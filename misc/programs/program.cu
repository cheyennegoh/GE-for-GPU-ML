#include "hip/hip_runtime.h"

#include <stdio.h>

void read_data(char *filename, float *data, size_t size)
{
	FILE *file = fopen(filename, "rb");
	fread(data, sizeof(float), size, file);
	fclose(file);
}

void write_data(char *filename, float *data, size_t size)
{
	FILE *file = fopen(filename, "wb");
	fwrite(data, sizeof(float), size, file);
	fclose(file);
}

__global__
void evaluate0(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 0;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (4 > 3)
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 8)
	r[0] = cosf(r[0]);
	if (4 > 9)
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[1])
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 5)
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > 5)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > r[1])
	r[0] = cosf(r[0]);
	if (3 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	if (r[1] > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 8)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 8)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 3)
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > 8)
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = cosf(r[0]);
	if (1 > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	if (6 > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 5)
	r[0] = cosf(r[0]);
	if (x[1] > 8)
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (8 > 6)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 3)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	if (5 > 8)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (4 > 9)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 5)
	r[0] = cosf(r[0]);
	if (r[0] > 7)
	r[0] = sinf(r[0]);
	if (4 > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > 9)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (7 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > 2)
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 6)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	if (3 > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > 8)
	r[0] = cosf(r[0]);
	if (4 > r[0])
	r[0] = cosf(r[0]);
	if (5 > 6)
	r[0] = sinf(r[0]);
	if (6 > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (1 > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate1(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 1;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (x[0] > 1)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (9 > r[0])
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 8)
	r[0] = cosf(r[0]);
	if (r[1] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > 2)
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 4)
	r[0] = sinf(r[0]);
	if (8 > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	if (3 > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > r[1])
	r[0] = sinf(r[0]);
	if (6 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > 3)
	r[0] = sinf(r[0]);
	if (7 > 3)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (9 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (3 > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 8)
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = sinf(r[0]);
	if (6 > 8)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (3 > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 8)
	r[0] = sinf(r[0]);
	if (x[0] > 3)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 5)
	r[0] = sinf(r[0]);
	if (4 > 2)
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > x[0])
	r[0] = cosf(r[0]);
	if (r[0] > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	if (9 > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	if (8 > x[0])
	r[0] = cosf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = cosf(r[0]);
	if (1 > r[1])
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > x[1])
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > 4)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > 8)
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (5 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > 1)
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	if (6 > 5)
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = sinf(r[0]);
	if (4 > r[0])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate2(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 2;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	if (3 > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 7)
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 1)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = sinf(r[0]);
	if (9 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > 3)
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > 4)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 7)
	r[0] = cosf(r[0]);
	if (3 > x[1])
	r[0] = sinf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 1)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > 3)
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 4)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 3)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 6)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > 5)
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > 1)
	r[0] = cosf(r[0]);
	if (9 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 9)
	r[0] = cosf(r[0]);
	if (r[0] > 1)
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = sinf(r[0]);
	if (1 > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > 8)
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > 3)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (3 > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = sinf(r[0]);
	if (r[1] > 7)
	r[0] = sinf(r[0]);
	if (8 > x[1])
	r[0] = cosf(r[0]);
	if (1 > 3)
	r[0] = cosf(r[0]);
	if (r[1] > 3)
	r[0] = cosf(r[0]);
	if (r[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > 3)
	r[0] = sinf(r[0]);
	if (1 > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > 6)
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate3(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 3;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	r[0] = sinf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 3)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > r[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (r[0] > 7)
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	if (4 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 6)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 9)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > 5)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > 7)
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (9 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > 1)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > 5)
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	if (5 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (7 > 8)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 4)
	r[0] = cosf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 3)
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (5 > r[0])
	r[0] = cosf(r[0]);
	if (7 > 9)
	r[0] = sinf(r[0]);
	if (r[0] > 7)
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (7 > 6)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (2 > 2)
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > 5)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 2)
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 5)
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > 8)
	r[0] = sinf(r[0]);
	if (x[1] > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 7)
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (4 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (6 > x[1])
	r[0] = cosf(r[0]);
	if (4 > 3)
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (5 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (5 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 8)
	r[0] = cosf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate4(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 4;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (x[0] > 6)
	r[0] = cosf(r[0]);
	if (2 > r[0])
	r[0] = sinf(r[0]);
	if (3 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > 9)
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (3 > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (9 > 3)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (5 > 8)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 1)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (5 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 2)
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 2)
	r[0] = sinf(r[0]);
	if (7 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > 1)
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 6)
	r[0] = sinf(r[0]);
	if (2 > 5)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (1 > 8)
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	if (6 > 9)
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = sinf(r[0]);
	if (8 > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > 5)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	if (6 > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 1)
	r[0] = sinf(r[0]);
	if (3 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 6)
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 1)
	r[0] = cosf(r[0]);
	if (2 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > 9)
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 1)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > r[1])
	r[0] = cosf(r[0]);
	if (7 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 8)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (3 > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > 4)
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (7 > 3)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > 1)
	r[0] = cosf(r[0]);
	if (8 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}

int main(int argc, char *argv[])
{
	float *x, *pred, *d_x, *d_pred;

	x = (float *)malloc(10 * 2 * sizeof(float));
	pred = (float *)malloc(5 * 10 * sizeof(float));

	if (argc > 1)
	{
		read_data(argv[1], (float *)x, 10 * 2);
	}

	hipMalloc(&d_x, 10 * 2 * sizeof(float));
	hipMalloc(&d_pred, 5 * 10 * sizeof(float));

	hipMemcpy(d_x, x, 10 * 2 * sizeof(float), hipMemcpyHostToDevice);

	evaluate0<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate1<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate2<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate3<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate4<<<((10 + 255) / 256), 256>>>(d_x, d_pred);

	hipMemcpy(pred, d_pred, 5 * 10 * sizeof(float), hipMemcpyDeviceToHost);

	if (argc > 2)
	{
		write_data(argv[2], (float *)pred, 5 * 10);
	}

	hipFree(d_x);
	hipFree(d_pred);

	free(x);
	free(pred);

	return 0;
}
