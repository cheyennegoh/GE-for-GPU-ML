#include "hip/hip_runtime.h"

#include <stdio.h>

void write_data(char *filename, float *data, size_t size)
{
	FILE *file = fopen(filename, "wb");
	fwrite(data, sizeof(float), size, file);
	fclose(file);
}

__global__
void evaluate0(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 0;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (4 > 3)
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 8)
	r[0] = cosf(r[0]);
	if (4 > 9)
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[1])
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 5)
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > 5)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > r[1])
	r[0] = cosf(r[0]);
	if (3 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	if (r[1] > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 8)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 8)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 3)
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > 8)
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = cosf(r[0]);
	if (1 > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	if (6 > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 5)
	r[0] = cosf(r[0]);
	if (x[1] > 8)
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (8 > 6)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 3)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	if (5 > 8)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (4 > 9)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 5)
	r[0] = cosf(r[0]);
	if (r[0] > 7)
	r[0] = sinf(r[0]);
	if (4 > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > 9)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (7 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > 2)
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 6)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	if (3 > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > 8)
	r[0] = cosf(r[0]);
	if (4 > r[0])
	r[0] = cosf(r[0]);
	if (5 > 6)
	r[0] = sinf(r[0]);
	if (6 > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (1 > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate1(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 1;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (x[0] > 1)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (9 > r[0])
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 8)
	r[0] = cosf(r[0]);
	if (r[1] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > 2)
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 4)
	r[0] = sinf(r[0]);
	if (8 > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	if (3 > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > r[1])
	r[0] = sinf(r[0]);
	if (6 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > 3)
	r[0] = sinf(r[0]);
	if (7 > 3)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (9 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (3 > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 8)
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = sinf(r[0]);
	if (6 > 8)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (3 > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 8)
	r[0] = sinf(r[0]);
	if (x[0] > 3)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 5)
	r[0] = sinf(r[0]);
	if (4 > 2)
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > x[0])
	r[0] = cosf(r[0]);
	if (r[0] > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	if (9 > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	if (8 > x[0])
	r[0] = cosf(r[0]);
	if (7 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = cosf(r[0]);
	if (1 > r[1])
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > x[1])
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > 4)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > 8)
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > 8)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (5 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > 1)
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	if (6 > 5)
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = sinf(r[0]);
	if (4 > r[0])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate2(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 2;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 9)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	if (3 > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 7)
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 1)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = sinf(r[0]);
	if (9 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > 3)
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (3 > 4)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > 7)
	r[0] = cosf(r[0]);
	if (3 > x[1])
	r[0] = sinf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 1)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > 3)
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 4)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 3)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 6)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 2)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > 5)
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > 1)
	r[0] = cosf(r[0]);
	if (9 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 9)
	r[0] = cosf(r[0]);
	if (r[0] > 1)
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = sinf(r[0]);
	if (1 > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > 8)
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > 3)
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (3 > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = sinf(r[0]);
	if (r[1] > 7)
	r[0] = sinf(r[0]);
	if (8 > x[1])
	r[0] = cosf(r[0]);
	if (1 > 3)
	r[0] = cosf(r[0]);
	if (r[1] > 3)
	r[0] = cosf(r[0]);
	if (r[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > 3)
	r[0] = sinf(r[0]);
	if (1 > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > 6)
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate3(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 3;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	r[0] = sinf(r[0]);
	if (x[0] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 3)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > r[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (r[0] > 7)
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	if (4 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 6)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 6)
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 9)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > 5)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (1 > 7)
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (9 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (7 > 1)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 2)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > 5)
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	if (5 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (7 > 8)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 4)
	r[0] = cosf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 3)
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = sinf(r[0]);
	if (x[1] > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 4)
	r[0] = cosf(r[0]);
	if (5 > r[0])
	r[0] = cosf(r[0]);
	if (7 > 9)
	r[0] = sinf(r[0]);
	if (r[0] > 7)
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 4)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (7 > 6)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (2 > 2)
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > 5)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (x[0] > 2)
	r[0] = cosf(r[0]);
	if (4 > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 5)
	r[0] = sinf(r[0]);
	if (r[1] > 5)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > 8)
	r[0] = sinf(r[0]);
	if (x[1] > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (4 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > 7)
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	if (4 > r[1])
	r[0] = sinf(r[0]);
	if (x[1] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (6 > x[1])
	r[0] = cosf(r[0]);
	if (4 > 3)
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	if (5 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (5 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > 8)
	r[0] = cosf(r[0]);
	if (r[0] > 6)
	r[0] = sinf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate4(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 4;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (x[0] > 6)
	r[0] = cosf(r[0]);
	if (2 > r[0])
	r[0] = sinf(r[0]);
	if (3 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > 9)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > 9)
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (3 > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (9 > 3)
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (5 > 8)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 9)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 1)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 6)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (9 > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (5 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > 3)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > 2)
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 2)
	r[0] = sinf(r[0]);
	if (7 > x[0])
	r[0] = cosf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > 1)
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 6)
	r[0] = sinf(r[0]);
	if (2 > 5)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > 1)
	r[0] = cosf(r[0]);
	if (3 > x[0])
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (5 > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (1 > 8)
	r[0] = sinf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (1 > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = sinf(r[0]);
	if (6 > 9)
	r[0] = cosf(r[0]);
	if (r[1] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (r[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > 7)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = sinf(r[0]);
	if (r[0] > 4)
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > r[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	if (5 > x[0])
	r[0] = sinf(r[0]);
	if (8 > 7)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > 5)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (6 > r[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > r[1])
	r[0] = cosf(r[0]);
	if (6 > x[0])
	r[0] = sinf(r[0]);
	if (r[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 1)
	r[0] = sinf(r[0]);
	if (3 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (8 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[0])
	r[0] = cosf(r[0]);
	if (r[1] > 1)
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > 6)
	r[0] = sinf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (3 > 1)
	r[0] = cosf(r[0]);
	if (2 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (7 > 9)
	r[0] = cosf(r[0]);
	if (r[0] > r[0])
	r[0] = cosf(r[0]);
	if (r[1] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 1)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	if (9 > r[0])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (2 > x[0])
	r[0] = sinf(r[0]);
	if (x[1] > x[1])
	r[0] = cosf(r[0]);
	if (x[1] > r[0])
	r[0] = cosf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (5 > r[1])
	r[0] = cosf(r[0]);
	if (7 > x[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (r[1] > x[1])
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (4 > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (r[0] > 8)
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > x[0])
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (3 > r[1])
	r[0] = cosf(r[0]);
	if (r[0] > x[0])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[1] > r[1])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > x[1])
	r[0] = cosf(r[0]);
	r[0] = cosf(r[0]);
	if (8 > 4)
	r[0] = cosf(r[0]);
	if (x[0] > r[1])
	r[0] = cosf(r[0]);
	if (7 > 3)
	r[0] = sinf(r[0]);
	r[0] = cosf(r[0]);
	if (9 > 1)
	r[0] = cosf(r[0]);
	if (8 > r[0])
	r[0] = cosf(r[0]);
	r[0] = sinf(r[0]);
	r[0] = sinf(r[0]);
	if (x[0] > r[0])
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}

int main(int argc, char *argv[])
{
	static float x[10][2] = {{1.8188397769118172, 1.2153098847303805}, {6.0, 3.6739403974420594e-16}, {-0.9375247682205794, 1.4031049707605447}, {1.2858791391047208e-15, -3.5}, {1.6742400165972686, 4.041972954736879}, {-1.23743686707646, 1.2374368670764564}, {4.001447509206, -2.673681746406834}, {-5.946010762444584, -1.1827350772227778}, {-3.0, -1.2858791391047208e-15}, {3.0036549212348937, 0.5974641111743921}};
	static float pred[5][10];

	float *d_x, *d_pred;

	hipMalloc(&d_x, 10 * 2 * sizeof(float));
	hipMalloc(&d_pred, 5 * 10 * sizeof(float));

	hipMemcpy(d_x, x, 10 * 2 * sizeof(float), hipMemcpyHostToDevice);

	evaluate0<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate1<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate2<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate3<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate4<<<((10 + 255) / 256), 256>>>(d_x, d_pred);

	hipMemcpy(pred, d_pred, 5 * 10 * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_pred);

	if (argc > 1)
	{
		write_data(argv[1], (float *)pred, 5 * 10);
	}

	return 0;
}
