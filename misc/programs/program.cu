#include "hip/hip_runtime.h"

#include <stdio.h>

void read_data(char *filename, float *data, size_t size)
{
	FILE *file = fopen(filename, "rb");
	fread(data, sizeof(float), size, file);
	fclose(file);
}

void write_data(char *filename, float *data, size_t size)
{
	FILE *file = fopen(filename, "wb");
	fwrite(data, sizeof(float), size, file);
	fclose(file);
}

__global__
void evaluate0(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 0;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (x[1] > 8)
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate1(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 1;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (4 > 8)
	r[0] = cosf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate2(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 2;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	r[0] = sinf(r[0]);
	if (8 > r[1])
	r[0] = sinf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate3(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 3;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (5 > x[1])
	r[0] = sinf(r[0]);
	if (6 > r[0])
	r[0] = sinf(r[0]);

	pred[tid] = r[0];
}
__global__
void evaluate4(float *x, float *pred)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= 10) return;

	x += 2 * tid;
	pred += 10 * 4;

	float r[2];
	for (int i = 0; i < 2; i++) r[i] = x[i % 2];

	if (r[1] > r[1])
	r[0] = cosf(r[0]);
	if (3 > 4)
	r[0] = sinf(r[0]);

	pred[tid] = r[0];
}

int main(int argc, char *argv[])
{
	float *x, *pred, *d_x, *d_pred;

	x = (float *)malloc(10 * 2 * sizeof(float));
	pred = (float *)malloc(5 * 10 * sizeof(float));

	if (argc > 1)
	{
		read_data(argv[1], (float *)x, 10 * 2);
	}

	hipMalloc(&d_x, 10 * 2 * sizeof(float));
	hipMalloc(&d_pred, 5 * 10 * sizeof(float));

	hipMemcpy(d_x, x, 10 * 2 * sizeof(float), hipMemcpyHostToDevice);

	evaluate0<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate1<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate2<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate3<<<((10 + 255) / 256), 256>>>(d_x, d_pred);
	evaluate4<<<((10 + 255) / 256), 256>>>(d_x, d_pred);

	hipMemcpy(pred, d_pred, 5 * 10 * sizeof(float), hipMemcpyDeviceToHost);

	if (argc > 2)
	{
		write_data(argv[2], (float *)pred, 5 * 10);
	}

	hipFree(d_x);
	hipFree(d_pred);

	free(x);
	free(pred);

	return 0;
}
